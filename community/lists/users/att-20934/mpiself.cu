#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void MPI_Warmup(){};
__global__ void MPI_Isend() {};
__global__ void MPI_Irecv() {};
__global__ void MPI_Wait() {};

int main(int argc, char** argv) {
  hipFree(0);  
  const int size=100000000;
  MPI_Init(&argc,&argv);

  int rank, procs;
  MPI_Status status;
  MPI_Request srequest, rrequest;

  MPI_Comm_size(MPI_COMM_WORLD,&procs);
  MPI_Comm_rank(MPI_COMM_WORLD,&rank);

  int *d_buf1, *d_buf2;
  hipMalloc((void**)&d_buf1,size*procs*sizeof(int));
  hipMalloc((void**)&d_buf2,size*procs*sizeof(int));

  int sleeptime=500;

  int neighbor=rank;

  //warm up mpi
  printf("Warmup\n");
  MPI_Warmup<<<1,1>>>();
  MPI_Allgather(d_buf1,size,MPI_INT,d_buf2,size,MPI_INT,MPI_COMM_WORLD);
  MPI_Warmup<<<1,1>>>();

  usleep(sleeptime);

  printf("Send/Receive Async\n");
  MPI_Isend<<<1,1>>>();
  MPI_Isend(d_buf1,size,MPI_INT,neighbor,0,MPI_COMM_WORLD,&srequest);
  MPI_Isend<<<1,1>>>();

  MPI_Irecv<<<1,1>>>();
  MPI_Irecv(d_buf2,size,MPI_INT,neighbor,0,MPI_COMM_WORLD,&rrequest);
  MPI_Irecv<<<1,1>>>();

  printf("MPI Wait\n");
  MPI_Wait<<<1,1>>>();
  MPI_Wait(&srequest,&status);
  MPI_Wait(&rrequest,&status);
  MPI_Wait<<<1,1>>>();
  usleep(sleeptime);
  
  hipFree(d_buf1);
  hipFree(d_buf2);

  MPI_Finalize();
}
