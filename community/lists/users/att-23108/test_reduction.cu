#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>
#include <mpi.h>

//#define USE_GPU

int main(int argc, char* argv[])
{
	int gpu_count;
	int current_gpu;

	MPI_Init(&argc, &argv);

	MPI_Comm_size(MPI_COMM_WORLD, &gpu_count);
	MPI_Comm_rank(MPI_COMM_WORLD, &current_gpu);

	double val = current_gpu;
	double* send_gpu;
	double* recv_gpu;
	hipMalloc((void**)&send_gpu, sizeof(double)*5);
	hipMalloc((void**)&recv_gpu, sizeof(double)*5);
	double* send_cpu = (double*)malloc(sizeof(double)*1);
	double* recv_cpu = (double*)malloc(sizeof(double)*1);

#ifdef USE_GPU
	hipMemcpy(send_gpu, &val, sizeof(double)*1,hipMemcpyHostToDevice);
	MPI_Reduce(send_gpu, recv_gpu,1,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
#else
	memcpy(send_cpu, &val, sizeof(double)*1);
	MPI_Reduce(send_cpu, recv_cpu,1,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
#endif

	if (current_gpu==0)
	{
#ifdef USE_GPU
		hipMemcpy(&val, recv_gpu, sizeof(double)*1,hipMemcpyDeviceToHost);
#else
		memcpy(&val, recv_cpu, sizeof(double)*1);
#endif
		printf("Sum of process ranks on GPU %d: %lf\n",current_gpu,val);
	}

	hipFree(send_gpu);
	hipFree(recv_gpu);
	free(send_cpu);
	free(recv_cpu);
	MPI_Finalize();
}
